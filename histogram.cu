#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <random>
#include <algorithm>
#include <vector>
#include <chrono>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

__global__ void histogram(float *d_data, int N, float *d_hist, int num_bins, float bin_width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        int bin = d_data[idx] / bin_width;
        atomicAdd(&d_hist[bin], 1.0f);
    }
}

int main()
{
    // Set up the random number generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // Generate random numbers
    const int N = 100'0000;
    float *d_data;
    hipMalloc(&d_data, N * sizeof(float));
    hiprandGenerateUniform(gen, d_data, N);

    // Set up the histogram
    const int num_bins = 32;
    float bin_width = 1.0f / num_bins;
    float *d_hist;
    hipMalloc(&d_hist, num_bins * sizeof(float));
    hipMemset(d_hist, 0, num_bins * sizeof(float));

    // Compute the histogram
    histogram<<<(N + 255) / 256, 256>>>(d_data, N, d_hist, num_bins, bin_width);

    // Copy the histogram back to the host
    float *h_hist = new float[num_bins];
    hipMemcpy(h_hist, d_hist, num_bins * sizeof(float), hipMemcpyDeviceToHost);

    // Print the histogram
    for (int i = 0; i < num_bins; i++)
    {
        std::cout << h_hist[i] << std::endl;
    }

    //create a vector to store the histogram
    std::vector<float> hist(h_hist, h_hist + num_bins);
    //find the maximum element in the histogram
    auto max = std::max_element(hist.begin(), hist.end());
    //find the index of the maximum element
    auto index = std::distance(hist.begin(), max);
    //print the index of the maximum element
    std::cout << "The index of the maximum element is: " << index << std::endl;
    //use cpu test algorithm run time
    std::vector<float> data(N);
    //random generate data use <random> library and <chrono> library to calculate the time
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0.0, 1.0);
    std::generate(data.begin(), data.end(), [&]() { return distribution(generator); });
    // Clean up
    delete[] h_hist;
    hipFree(d_hist);
    hipFree(d_data);
    hiprandDestroyGenerator(gen);
    return 0;
}