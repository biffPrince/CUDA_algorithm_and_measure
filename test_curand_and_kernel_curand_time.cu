#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
// curand in device API
__global__ void init(unsigned int seed, hiprandState_t* states) 
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &states[id]);
}
// use XORWOW algorithm to generate random number  with uniform distribution.
__global__ void generate(hiprandState_t* states, float* numbers) 
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    numbers[id] = hiprand_uniform(&states[id]);
}


int main()
{
    float *d_A;
    const int size= 1000'0000;
    // malloc device memory
    hipMalloc(&d_A,size*sizeof(float));
    
    //use cuda_runtime measure curand in host API;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); //start record time
    //generate random number
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,2233ULL); //set seed
    hiprandGenerateUniform(gen,d_A,size);
    hipEventRecord(stop); //stop record time
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("Time to generate random number in host API: %3.1f ms\n",elapsedTime);

    //use cuda_runtime measure curand in device API;
    hiprandState_t* states;
    hipMalloc(&states,size*sizeof(hiprandState_t));
    hipEventRecord(start); //start record time
    init<<<size/256,256>>>(2237,states);
    generate<<<size/256,256>>>(states,d_A);
    hipEventRecord(stop); //stop record time
    hipEventSynchronize(stop);
    float elapsedTimeDevice;
    hipEventElapsedTime(&elapsedTimeDevice,start,stop);
    printf("Time to generate random number in device API: %3.1f ms\n",elapsedTimeDevice);

    //free memory
    hipFree(d_A);
    hipFree(states);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hiprandDestroyGenerator(gen);
    return 0;
}