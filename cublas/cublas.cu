#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>


// it can be optimaized with grid-stride loop.
__global__ void vectorAdd(int n, float *x, float *y,float *z)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        // z[i] =  x[i] + y[i];
        z[i]=__fadd_rn(x[i],y[i]);
}

__global__ void compareVectorsKernel(float* d, float* b, int* areEqual, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n && (fabs(d[index] - b[index]) > 1e-6)){
        *areEqual = 0;
        return ;
    }
}

int main()
{
    const int n = 1000'0000;
    //all is device memory
    float *a, *b,*d ;
    hipMalloc(&a, n * sizeof(float));
    hipMalloc(&b, n * sizeof(float));
    hipMalloc(&d, n * sizeof(float));
    //use kernel function and measure it
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Set up the random number generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 123ULL);
    hiprandGenerateUniform(gen,a,n);
    hiprandGenerateUniform(gen,b,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millisecondsR = 0;
    hipEventElapsedTime(&millisecondsR, start, stop);
    printf("The elapsed time of random in microseconds is %f\n", millisecondsR*1000);
    hipEventRecord(start);
    vectorAdd<<<(n + 255) / 256, 256>>>(n, a, b, d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millisecondsK = 0;
    hipEventElapsedTime(&millisecondsK, start, stop);
    // cudaEventDestroy(startK);
    // cudaEventDestroy(stopK);
    printf("The elapsed time of kernel in microseconds is %f\n", millisecondsK*1000);
    //cuBLAS handle
    hipblasHandle_t cuHandle;
    //use cuda runtime measure the time
    
    // cudaEvent_t start, stop;
    // cudaEventCreate(&start);
    // cudaEventCreate(&stop); 

    hipblasCreate(&cuHandle);
    float alpha = 1.0;
    hipEventRecord(start);
    hipblasSaxpy(cuHandle, n, &alpha, a, 1, b, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("The elapsed time of cublas in microseconds is %f\n", milliseconds*1000);


    //check the result  of kernel. but b and d is device memory, need kernel function do that. 
    //use kernel function to compare the result of kernel and cublas

    //measure the time of compare kernel function
    hipEventRecord(start);
    int * areEqual;
    hipMalloc(&areEqual, sizeof(int));
    compareVectorsKernel<<<(n + 255) / 256, 256>>>(d, b, areEqual, n);
    hipDeviceSynchronize();
    int h_areEqual;
    hipMemcpy(&h_areEqual, areEqual, sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millisecondsC = 0;
    hipEventElapsedTime(&millisecondsC, start, stop);
    printf("The elapsed time of compare kernel in microseconds is %f\n", millisecondsC*1000);
    if (!h_areEqual) {
        printf("The kernel and cublas results are equal\n");
    } else {
        printf("The kernel and cublas results are not equal\n");
    }
    // create two host vector to store the result of kernel and cublas
    // measure the time of compare in CPU and move the result of kernel and cublas to host memory.
    hipEventRecord(start);
    float *h_d = (float*)malloc(n * sizeof(float));
    float *h_b = (float*)malloc(n * sizeof(float));
    hipMemcpy(h_d, d, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_b, b, n * sizeof(float), hipMemcpyDeviceToHost);
    bool state=false;
    for (int i = 0; i < n; i++) {
        if (h_d[i] != h_b[i]) {
            state = true;
            break;
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millisecondsH = 0;
    hipEventElapsedTime(&millisecondsH, start, stop);
    printf("The elapsed time of compare in CPU in microseconds is %f\n", millisecondsH*1000);
    if(state)
    {
        printf("The kernel and cublas results are not equal\n");
    }else
    {
        printf("The kernel and cublas results are equal\n");
    }
    hiprandDestroyGenerator(gen);
    printf("program reach here %d\n",__LINE__);
    hipEventDestroy(start);
    hipEventDestroy(stop);   
    hipblasDestroy(cuHandle);
    hipFree(a);
    hipFree(b);
    hipFree(d);
    free(h_d);
    free(h_b);
    return 0;
}